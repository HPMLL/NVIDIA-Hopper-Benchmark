#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <mma.h>
#include <stdio.h>
#include <type_traits>
#include <stdlib.h>

#include "../../include/util.h"

#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif 

#ifndef ILP
#define ILP 4
#endif 

#ifndef ITERs
#define ITERs 10000000
#endif

#define H2D  hipMemcpyHostToDevice
#define D2H  hipMemcpyDeviceToHost

#define StrideA 16*4
#define StrideB 8*8
#define StrideC 16*8

using u32 = uint32_t;
using u64 = uint64_t;
using f16 = half;
using f32 = float;
using i8 = int8_t;
using i32 = int32_t;

#define mmasp_inst_tf32_f32(ii2,ii4) \
{\
    asm volatile(\
        "mma.sp.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6,%7}, {%8,%9,%10,%11},%12,0x0;\n" \
        : "=f"(D[ii4]), "=f"(D[ii4+1]), "=f"(D[ii4+2]), "=f"(D[ii4+3]) \
        :   \
        "r"(a_frag[ii2]), "r"(a_frag[ii2+1]), \
        "r"(b_frag[ii2]), "r"(b_frag[ii2+1]), \
        "f"(c_frag[ii4]), "f"(c_frag[ii4+1]), "f"(c_frag[ii4+2]), "f"(c_frag[ii4+3]), \
        "r"(e_frag) \
    ); \
}

__global__ void  warm_up() {
    printf("warm up\n");
}


template<class T1, class T2>
__global__  void bench(T1* A, T1* B, T2* C, char *indics, u64 *start_clk, u64 *end_clk) {
    u64 start, end;

    u32 bid = blockIdx.x;
    u32 bdim = blockDim.x;
    u32 tid = threadIdx.x;
    u32 gtid = bid * bdim + tid;
    u32 gwid = gtid / 32;

    T1 a_frag_float[2 * ILP];
    T1 b_frag_float[2 * ILP];
    T2 c_frag_float[4 * ILP];
    u32 e_frag = reinterpret_cast<const u32*>(&indics[0])[gwid];

    for (int ii = 0; ii < 2 * ILP; ii++) {
        a_frag_float[ii] = A[gwid*StrideA+ii];
    }
    for (int ii = 0; ii < 2 * ILP; ii++) {
        b_frag_float[ii] = B[gwid*StrideB+ii];
    }
    for (int ii = 0; ii < 4 * ILP; ii++) {
        c_frag_float[ii] = 0.0f;
    }

    if (std::is_same<T2, f32>::value && std::is_same<T1, f32>::value) {
        const u32* a_frag = reinterpret_cast<const u32*>(&a_frag_float[0]);
        const u32* b_frag = reinterpret_cast<const u32*>(&b_frag_float[0]);
        f32* c_frag = reinterpret_cast<f32*>(&c_frag_float[0]);
        f32* D = reinterpret_cast<f32*>(&c_frag_float[0]);
        __syncthreads();
        asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory"); 
        for (int ii = 0; ii < ITERs; ii++) {
#pragma unroll
            for (int inst = 0; inst < ILP; inst++) {
                mmasp_inst_tf32_f32(inst*2, inst*4);
            }
            __syncwarp();
        }
        __syncthreads();
    } else assert(0);

    asm volatile("mov.u64 %0, %%clock64;" : "=l"(end)::"memory");

    for (int ii = 0; ii < ILP * 4; ii++) {
        C[gwid * StrideC + ii] = c_frag_float[ii];
    }

    start_clk[gtid] = start;
    end_clk[gtid] = end;
}

template<class T1, class T2>
void run(u32 threads_per_block) {
    u32 warp_num = threads_per_block / WARP_SIZE;
    dim3 grid_dim = SMs;
    dim3 block_dim = threads_per_block;

    // A 16 * 16; B 16 * 8; C 16 * 8 each warp
    T1 *data_Ad, *data_Bd;
    T2 *data_Cd;
    T1 *data_Ah, *data_Bh;
    T2 *data_Ch;
    u64 *start_clkd, *end_clkd;
    u64 *start_clkh, *end_clkh, *clk;
    char *indics_host, *indics_device;

    u32 sizeA = StrideA * warp_num * sizeof(T1) * SMs;
    u32 sizeB = StrideB * warp_num * sizeof(T1) * SMs;
    u32 sizeC = StrideC * warp_num * sizeof(T2) * SMs;
    u32 sizeclk = threads_per_block * SMs * sizeof(u64);
    u32 sizee = StrideA * warp_num * sizeof(char) * SMs; //surplus

    data_Ah = reinterpret_cast<T1*>(malloc(sizeA));
    data_Bh = reinterpret_cast<T1*>(malloc(sizeB));
    data_Ch = reinterpret_cast<T2*>(malloc(sizeC));
    start_clkh = reinterpret_cast<u64*>(malloc(sizeclk));
    end_clkh = reinterpret_cast<u64*>(malloc(sizeclk));
    clk = reinterpret_cast<u64*>(malloc(sizeclk));
    indics_host = reinterpret_cast<char *>(malloc(sizee));

    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Ad), sizeA));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Bd), sizeB));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Cd), sizeC));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&start_clkd), sizeclk));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&end_clkd), sizeclk));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&indics_device), sizee));

    for (u32 ii = 0; ii < StrideA; ii++) {
        data_Ah[ii] = ii + 1;
    }
    for (u32 ii = 0; ii < StrideB; ii++) {
        data_Bh[ii] = ii + 1;
    }
    for (u32 ii = 0; ii < StrideC; ii++) {
        data_Ch[ii] = ii + 2;
    }

    if (std::is_same<T1,f32>::value) {
        init_indics(indics_host, sizee, 4);
    } else init_indics(indics_host, sizee, 2);

    gpuErrchk(hipMemcpy(data_Ad, data_Ah, sizeA, H2D));
    gpuErrchk(hipMemcpy(data_Bd, data_Bh, sizeB, H2D));
    gpuErrchk(hipMemcpy(data_Cd, data_Ch, sizeC, H2D));
    gpuErrchk(hipMemcpy(indics_device, indics_host, sizee, H2D));

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    bench<<<grid_dim, block_dim>>>(data_Ad, data_Bd, data_Cd, indics_device, start_clkd, end_clkd);

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipEventRecord(end);
    hipEventSynchronize(end);

    f32 time;
    hipEventElapsedTime(&time, start, end);
    
    std::cout << "Time : " << time << " ms" << std::endl;
    std::cout << "Throughput(whole kernel) : " << f32(16) * 8 * 8 * 2 * warp_num * SMs * ILP / 1e9 * ITERs / time << " TFLOPS/TOPS" << std::endl;

    gpuErrchk(hipMemcpy(start_clkh, start_clkd, sizeclk, D2H));
    gpuErrchk(hipMemcpy(end_clkh, end_clkd, sizeclk, D2H));

    for (int ii = 0; ii < sizeclk/sizeof(u64); ii++) {
        clk[ii] = end_clkh[ii] - start_clkh[ii];
    }

    u64 clock_latency = 
              *std::max_element(clk, clk+threads_per_block * SMs);


    std::cout << "Latency = "  << clock_latency << std::endl;
    std::cout << "Throughput(mma inst) = " << f32(16) * 8 * 8 * 2 * warp_num * SMs * ILP / 1e12 * GPUFreq * ITERs / clock_latency << " TFLOPS/TOPS" <<std::endl;

    hipFree(data_Ad);
    hipFree(data_Bd);
    hipFree(data_Cd);
    hipFree(start_clkd);
    hipFree(end_clkd);
    hipFree(indics_device);
}

int main() {
    InitGPUDeviceProperty();
    warm_up<<<1, 1>>>();
    hipDeviceSynchronize();

    std::cout << "**********START MMASP TF32 F32 BENCH***********" << std::endl;
    for (u32 threads_per_block = 32; threads_per_block <= 1024; threads_per_block*=2) {
        // 32 64 128 256 512 1024
        std::cout << "mma.sp.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 TEST with ILP = " << ILP << ", warp num = " << threads_per_block / WARP_SIZE << ", SMs = " << SMs << ", GPUFreq = " << GPUFreq << std::endl;
        run<f32, f32>(threads_per_block);
    }
}