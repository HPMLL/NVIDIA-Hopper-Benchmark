#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <cstdint>
#include <cstdio>
#include <hip/hip_fp16.h>
#include "mma_sm90_gmma.hpp"
using namespace cute;

constexpr int M = 64;
constexpr int K = 8;

template <int N>
struct GMMA_Selector_SS;

template <>
struct GMMA_Selector_SS<256> {
    using type = SM90_64x256x8_F32TF32TF32_SS_TN<>;
};
template <>
struct GMMA_Selector_SS<128> {
    using type = SM90_64x128x8_F32TF32TF32_SS_TN<>;
};
template <>
struct GMMA_Selector_SS<64> {
    using type = SM90_64x64x8_F32TF32TF32_SS_TN<>;
};
template <>
struct GMMA_Selector_SS<32> {
    using type = SM90_64x32x8_F32TF32TF32_SS_TN<>;
};
template <>
struct GMMA_Selector_SS<16> {
    using type = SM90_64x16x8_F32TF32TF32_SS_TN<>;
};
template <>
struct GMMA_Selector_SS<8> {
    using type = SM90_64x8x8_F32TF32TF32_SS_TN<>;
};
template <int N>
struct GMMA_Selector_RS;

template <>
struct GMMA_Selector_RS<256> {
    using type = SM90_64x256x8_F32TF32TF32_RS_TN<>;
};
template <>
struct GMMA_Selector_RS<128> {
    using type = SM90_64x128x8_F32TF32TF32_RS_TN<>;
};
template <>
struct GMMA_Selector_RS<64> {
    using type = SM90_64x64x8_F32TF32TF32_RS_TN<>;
};
template <>
struct GMMA_Selector_RS<32> {
    using type = SM90_64x32x8_F32TF32TF32_RS_TN<>;
};
template <>
struct GMMA_Selector_RS<16> {
    using type = SM90_64x16x8_F32TF32TF32_RS_TN<>;
};
template <>
struct GMMA_Selector_RS<8> {
    using type = SM90_64x8x8_F32TF32TF32_RS_TN<>;
};


template <int N>
__global__ void
wgmma_m64nNk8_tf32_SS(float *gm_d, float *gm_a, float *gm_b,         
                                      float *gm_c, uint8_t sm_layout, int exe_time) {                        
  constexpr int RegCount = 64 * N / 128;
  extern __shared__ char shem[];                                                  
  using GMMA_t = typename GMMA_Selector_SS<N>::type;  
  GMMA_t gmma_instance;
  typename GMMA_t::CRegisters reg_d;                                                         
  typename GMMA_t::ARegisters reg_a;     
  
  
  float *shem_a = (float *)shem;                                               
  for (int i = threadIdx.x; i < M * K; i += blockDim.x) {                        
    shem_a[i] = gm_a[i];                                                               
  }                                                                           

  float *shem_b = (float *)(shem + sizeof(float) * M * K);                
  for (int i = threadIdx.x; i < N * K; i += blockDim.x) {                        
    shem_b[i] = gm_b[i];                                                               
  }                                                                                  
                                                                                     
  for (int i = 0; i < sizeof(reg_d)/sizeof(float); ++i) {                            
    reg_d[i] = gm_c[threadIdx.x + blockDim.x * i];                                   
  }                                                                                  
                                                                                     
  __syncthreads();                                                                   
                                                                                     
  uint32_t sm_a_addr = static_cast<uint32_t>(__cvta_generic_to_shared(shem_a));        
  uint32_t sm_b_addr = static_cast<uint32_t>(__cvta_generic_to_shared(shem_b));        
                                                                                     
  GmmaDescriptor desc_a, desc_b;                                                     
  desc_a.layout_type_ = sm_layout;                                                   
  desc_b.layout_type_ = sm_layout;                                                   
                                                                                     
  desc_a.start_address_ = sm_a_addr >> 4;                                            
  desc_b.start_address_ = sm_b_addr >> 4;                                            
                                                                                     
  desc_a.base_offset_ = 0;                                                           
  desc_b.base_offset_ = 0;                                                           
                                                                                     
  desc_a.leading_byte_offset_ = (8 * 4 * sizeof(float)) >> 4;                       
  desc_b.leading_byte_offset_ = (8 * 4 * sizeof(float)) >> 4;                       
                                                                                     
  desc_a.stride_byte_offset_ = (2 * 8 * 4 * sizeof(float)) >> 4;                    
  desc_b.stride_byte_offset_ = (2 * 8 * 4 * sizeof(float)) >> 4;                    
                                                                                     
  reg_a[0] = desc_a.desc_;                                                                                         
  for (uint64_t i = 0; i < exe_time; ++i) {                  
    callFmaWithRegAD<GMMA_t, 1, RegCount>(gmma_instance, reg_a, std::make_index_sequence<1>{}, desc_b.desc_, reg_d, std::make_index_sequence<RegCount>{});
  }

  asm volatile("wgmma.commit_group.sync.aligned;\n" ::: "memory");
  asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(0) : "memory");

  for (int i = 0; i < sizeof(reg_d)/sizeof(float); ++i) {
    gm_d[threadIdx.x + blockDim.x * i] = reg_d[i];
  }
}
template <int N>
void test_m64nNk8_tf32_SS(std::string init_method) {
  int mat_a_size = M * K;
  int mat_b_size = N * K;
  int mat_c_size = M * N;

  float *mat_a_host = new float[mat_a_size];
  fill_mat(mat_a_host, mat_a_size, init_method);

  float *mat_b_host = new float[mat_b_size];
  fill_mat(mat_b_host, mat_b_size, init_method);

  float *mat_c_host = new float[mat_c_size];
  fill_mat(mat_c_host, mat_c_size, init_method);

  float *mat_d_host = new float[mat_c_size];
  fill_mat(mat_d_host, mat_c_size, init_method);

  float *mat_a_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_a_dev, mat_a_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_a_dev, mat_a_host, mat_a_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_b_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_b_dev, mat_b_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_b_dev, mat_b_host, mat_b_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_c_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_c_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_c_dev, mat_c_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_d_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_d_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_d_dev, mat_d_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  int dyn_shared_size =
      mat_a_size * sizeof(float) + mat_b_size * sizeof(float);

  hipFuncSetAttribute(reinterpret_cast<const void*>(wgmma_m64nNk8_tf32_SS<N>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       dyn_shared_size);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int num_sm = prop.multiProcessorCount;
  
  GpuTimer timer;
  timer.Start();
  wgmma_m64nNk8_tf32_SS<N><<<num_sm, 128, dyn_shared_size>>>(
        mat_d_dev, mat_a_dev, mat_b_dev, mat_c_dev, 0, 100000);
  timer.Stop();
  float elapsed_time = timer.Elapsed();

  double FLOPS = (double)2 * M * K * N * 100000 * num_sm;
  double TFLOPS = FLOPS / elapsed_time / 1000 / 1000 / 1000;
  std::cout << "SM90_64xNx8_F32TF32TF32_SS_TN " << init_method
	      << " A in shem "
	      << "M=" << M << ","
	      << "N=" << N << ","
	      << "K=" << K << "  elapsed_time: " << elapsed_time << "ms " << TFLOPS << "TFLOPS"
        << std::endl;
}

template <int N>
__global__ void
wgmma_m64nNk8_tf32_RS(float *gm_d, float *gm_a, float *gm_b,         
                                      float *gm_c, uint8_t sm_layout, int exe_time) {                        
  constexpr int RegCount = 64 * N / 128;    
  extern __shared__ char shem[];                                
  using GMMA_t = typename GMMA_Selector_RS<N>::type;  
  GMMA_t gmma_instance;
  typename GMMA_t::CRegisters reg_d;                                                         
  typename GMMA_t::ARegisters reg_a;   

  float *shem_a = (float *)shem;                                               
  for (int i = threadIdx.x; i < M * K; i += blockDim.x) {                        
    shem_a[i] = gm_a[i];
  }                                                                                  
                                                                                     
  float *shem_b = (float *)(shem + sizeof(float) * M * K);                
                                                                                     
  for (int i = threadIdx.x; i < N * K; i += blockDim.x) {                        
    shem_b[i] = gm_b[i];                                                               
  }                                                                                  
                                                                                     
  for (int i = 0; i < sizeof(reg_d)/sizeof(float); ++i) {                            
    reg_d[i] = gm_c[threadIdx.x + blockDim.x * i];                                   
  }                                                                                  
                                                                                     
  __syncthreads();                                                                   

  for (int i = 0; i < sizeof(reg_a)/sizeof(float); i += 1) {                        
    reinterpret_cast<float&>(reg_a[i]) = shem_a[i];
  }                                                                                  
                                                                                     
  uint32_t sm_a_addr = static_cast<uint32_t>(__cvta_generic_to_shared(shem_a));        
  uint32_t sm_b_addr = static_cast<uint32_t>(__cvta_generic_to_shared(shem_b));        
                                                                                     
  GmmaDescriptor desc_b;                                                     
  desc_b.layout_type_ = sm_layout;                                                   
                                                                                     
  desc_b.start_address_ = sm_b_addr >> 4;                                            
                                                                                     
  desc_b.base_offset_ = 0;                                                           
                                                                                     
  desc_b.leading_byte_offset_ = (8 * 4 * sizeof(float)) >> 4;                       
                                                                                     
  desc_b.stride_byte_offset_ = (2 * 8 * 4 * sizeof(float)) >> 4;                    
                                                                                     
  for (uint64_t i = 0; i < exe_time; ++i) {                  
    callFmaWithRegAD<GMMA_t, 4, RegCount>(gmma_instance, reg_a, std::make_index_sequence<4>{}, desc_b.desc_, reg_d, std::make_index_sequence<RegCount>{});
  }

  asm volatile("wgmma.commit_group.sync.aligned;\n" ::: "memory");
  asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(0) : "memory");

  for (int i = 0; i < sizeof(reg_d)/sizeof(float); ++i) {
    gm_d[threadIdx.x + blockDim.x * i] = reg_d[i];
  }
}



template <int N>
void test_m64nNk8_tf32_RS(std::string init_method) {
  int mat_a_size = M * K;
  int mat_b_size = N * K;
  int mat_c_size = M * N;

  float *mat_a_host = new float[mat_a_size];
  fill_mat(mat_a_host, mat_a_size, init_method);

  float *mat_b_host = new float[mat_b_size];
  fill_mat(mat_b_host, mat_b_size, init_method);

  float *mat_c_host = new float[mat_c_size];
  fill_mat(mat_c_host, mat_c_size, init_method);

  float *mat_d_host = new float[mat_c_size];
  fill_mat(mat_d_host, mat_c_size, init_method);

  float *mat_a_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_a_dev, mat_a_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_a_dev, mat_a_host, mat_a_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_b_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_b_dev, mat_b_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_b_dev, mat_b_host, mat_b_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_c_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_c_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_c_dev, mat_c_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_d_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_d_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_d_dev, mat_d_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  int dyn_shared_size =
      mat_a_size * sizeof(float) + mat_b_size * sizeof(float);

  hipFuncSetAttribute(reinterpret_cast<const void*>(wgmma_m64nNk8_tf32_RS<N>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       dyn_shared_size);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int num_sm = prop.multiProcessorCount;
  
  GpuTimer timer;
  timer.Start();
  wgmma_m64nNk8_tf32_RS<N><<<num_sm, 128, dyn_shared_size>>>(
        mat_d_dev, mat_a_dev, mat_b_dev, mat_c_dev, 0, 100000);
  timer.Stop();
  float elapsed_time = timer.Elapsed();

  double FLOPS = (double)2 * M * K * N * 100000 * num_sm;
  double TFLOPS = FLOPS / elapsed_time / 1000 / 1000 / 1000;

  std::cout << "SM90_64xNx8_F32TF32TF32_RS_TN " << init_method
	      << " A in reg "
	      << "M=" << M << ","
	      << "N=" << N << ","
	      << "K=" << K << "  elapsed_time: " << elapsed_time << "ms " << TFLOPS << "TFLOPS"
        << std::endl;
}



int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << " Usage ./tf32 <random/zero> <8 16 32 64 128 256> <ss/rs>" << std::endl;
    return -1;
  }
  std::string init_method = std::string(argv[1]);
  int N = std::stoi(argv[2]);
  std::string a_scope = std::string(argv[3]);
  if (a_scope == "ss") {
    if (N == 256) {
      test_m64nNk8_tf32_SS<256>(init_method);
    } else if (N == 128) {
      test_m64nNk8_tf32_SS<128>(init_method);
    } else if (N == 64) {
      test_m64nNk8_tf32_SS<64>(init_method);
    } else if (N == 32) {
      test_m64nNk8_tf32_SS<32>(init_method);
    } else if (N == 16) {
      test_m64nNk8_tf32_SS<16>(init_method);
    } else if (N == 8) {
      test_m64nNk8_tf32_SS<8>(init_method);
    } else {
      std::cout << "Unimplemented value of N: " << N << std::endl;
    }
  } else if (a_scope == "rs") {
    if (N == 256) {
      test_m64nNk8_tf32_RS<256>(init_method);
    } else if (N == 128) {
      test_m64nNk8_tf32_RS<128>(init_method);
    } else if (N == 64) {
      test_m64nNk8_tf32_RS<64>(init_method);
    } else if (N == 32) {
      test_m64nNk8_tf32_RS<32>(init_method);
    } else if (N == 16) {
      test_m64nNk8_tf32_RS<16>(init_method);
    } else if (N == 8) {
      test_m64nNk8_tf32_RS<8>(init_method);
    } else {
      std::cout << "Unimplemented value of N: " << N << std::endl;
    }
  }
}


