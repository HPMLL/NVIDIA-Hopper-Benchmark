#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <type_traits>
#include <stdlib.h>
#include <stdio.h>

#include "../../include/util.h"

#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif 

#ifndef ILP
#define ILP 4
#endif 

#ifndef ITERs
#define ITERs 10000
#endif

#define H2D  hipMemcpyHostToDevice
#define D2H  hipMemcpyDeviceToHost

#define StrideA 16*4
#define StrideB 8*4
#define StrideC 16*8

using u32 = uint32_t;
using u64 = uint64_t;
using f16 = half;
using f32 = float;
using i8 = int8_t;
using i32 = int32_t;

#define mma_inst_tf32_f32(ii1,ii2,ii4) \
{\
    asm volatile(\
        "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n" \
        : "=f"(D[ii4]), "=f"(D[ii4+1]), "=f"(D[ii4+2]), "=f"(D[ii4+3]) \
        :   \
        "r"(a_frag[ii2]), "r"(a_frag[ii2+1]), \
        "r"(b_frag[ii1]), \
        "f"(c_frag[ii4]), "f"(c_frag[ii4+1]), "f"(c_frag[ii4+2]), "f"(c_frag[ii4+3])\
    ); \
}

__global__ void  warm_up() {
    printf("warm up\n");
}

template<class T1, class T2>
__global__  void bench(T1* A, T1* B, T2* C, u64* clk_s, u64* clk_e) {
    u64 start, end;
    u32 tid = threadIdx.x;
    u32 wid = tid / 32;

    T1 a_frag_float[2 * ILP];
    T1 b_frag_float[1 * ILP];
    T2 c_frag_float[4 * ILP];

    for (int ii = 0; ii < 2 * ILP; ii++) {
        a_frag_float[ii] = A[wid*StrideA+ii];
    }
    for (int ii = 0; ii < 1 * ILP; ii++) {
        b_frag_float[ii] = B[wid*StrideB+ii];
    }
    for (int ii = 0; ii < 4 * ILP; ii++) {
        c_frag_float[ii] = 0.0f;
    }

    if (std::is_same<T2, f32>::value && std::is_same<T1, f32>::value) {
        const u32* a_frag = reinterpret_cast<const u32*>(&a_frag_float[0]);
        const u32* b_frag = reinterpret_cast<const u32*>(&b_frag_float[0]);
        f32* c_frag = reinterpret_cast<f32*>(&c_frag_float[0]);
        f32* D = reinterpret_cast<f32*>(&c_frag_float[0]);
        __syncthreads();
        asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory"); 
        for (int ii = 0; ii < ITERs; ii++) {
#pragma unroll
            for (int inst = 0; inst < ILP; inst++) {
                mma_inst_tf32_f32(inst, inst*2, inst*4);
            }
            __syncwarp();
        }
        __syncthreads();
    } else assert(0);

    asm volatile("mov.u64 %0, %%clock64;" : "=l"(end)::"memory");
    clk_s[tid] = start;
    clk_e[tid] = end;

    for (int ii = 0; ii < ILP * 4; ii++) {
        C[wid * StrideC + ii] = c_frag_float[ii];
    }

}

template<class T1, class T2>
void run(u32 threads_num) {
    // threads_num is always divisble by WARP_SIZE
    u32 warp_num = threads_num / WARP_SIZE;
    dim3 grid_dim = 1;
    dim3 block_dim = threads_num;

    // A 16 * 8; B 8 * 8; C 16 * 8 each warp
    T1 *data_Ad, *data_Bd;
    T2 *data_Cd;
    T1 *data_Ah, *data_Bh;
    T2 *data_Ch;
    u64 *clk_start_host, *clk_end_host;
    u64 *clk_start_device, *clk_end_device;

    u32 sizeA = StrideA * warp_num * sizeof(T1) * SMs;
    u32 sizeB = StrideB * warp_num * sizeof(T1) * SMs;
    u32 sizeC = StrideC * warp_num * sizeof(T2) * SMs;
    u32 sizeclk = threads_num * sizeof(u64);

    data_Ah = reinterpret_cast<T1*>(malloc(sizeA));
    data_Bh = reinterpret_cast<T1*>(malloc(sizeB));
    data_Ch = reinterpret_cast<T2*>(malloc(sizeC));
    clk_start_host = reinterpret_cast<u64*>(malloc(sizeclk));
    clk_end_host = reinterpret_cast<u64*>(malloc(sizeclk));

    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Ad), sizeA));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Bd), sizeB));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&data_Cd), sizeC));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&clk_start_device), sizeclk));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&clk_end_device), sizeclk));

    for (u32 ii = 0; ii < StrideA; ii++) {
        data_Ah[ii] = ii + 1;
    }
    for (u32 ii = 0; ii < StrideB; ii++) {
        data_Bh[ii] = ii + 1;
    }
    for (u32 ii = 0; ii < StrideC; ii++) {
        data_Ch[ii] = ii + 2;
    }

    gpuErrchk(hipMemcpy(data_Ad, data_Ah, sizeA, H2D));
    gpuErrchk(hipMemcpy(data_Bd, data_Bh, sizeB, H2D));
    gpuErrchk(hipMemcpy(data_Cd, data_Ch, sizeC, H2D));

    bench<<<grid_dim, block_dim>>>(data_Ad, data_Bd, data_Cd, clk_start_device, clk_end_device);

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(clk_start_host, clk_start_device, sizeclk, D2H));
    gpuErrchk(hipMemcpy(clk_end_host, clk_end_device, sizeclk, D2H));
    
    u32 clock = *std::max_element(clk_end_host, clk_end_host+threads_num) - 
                *std::min_element(clk_start_host, clk_start_host+threads_num);

    std::cout << "Latency = " << double(clock) / ITERs << std::endl
              << "Throughput = " << float(ILP * 16 * 8 * 4 * warp_num * ITERs) / clock << " FMA/SM/clk, ideally " 
              << float(ILP / 1e6 * 16 * 8 * 4 * warp_num * ITERs * 2) / clock * GPUFreq * SMs / 1e6 << " TFLOPS/TOPS" << std::endl;

    hipFree(data_Ad);
    hipFree(data_Bd);
    hipFree(data_Cd);
    hipFree(clk_start_device);
    hipFree(clk_end_device);
}

int main() {
    InitGPUDeviceProperty();
    warm_up<<<1, 1>>>();
    hipDeviceSynchronize();

    std::cout << "**********START MMA TF32 F32 BENCH***********" << std::endl;
    for (u32 threads_num = 32; threads_num <= 1024; threads_num*=2) {
        // 32 64 128 256 512 1024
        std::cout << "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 TEST with ILP = " << ILP << ", warp num = " << threads_num / WARP_SIZE << ", SMs = " << SMs << ", GPUFreq = " << GPUFreq << std::endl;
        run<f32, f32>(threads_num);
    }
}

