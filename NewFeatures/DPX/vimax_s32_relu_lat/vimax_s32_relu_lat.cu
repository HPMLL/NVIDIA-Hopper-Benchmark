#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1
#define THREADS_PER_SM 1
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 1024

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	register T s1 = data1[gid];
	register T s2 = data2[gid]+1;
	register T s3 = s1 + 2;
	register T s4 = s2;
	register T result = 1;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {

 		s1 = __vimax_s32_relu(s1, s2);

	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	result = s1 + s2 + s3 + s4;
	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = result;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	int32_t *data1 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *data2 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *res = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	int32_t *data1_g;
	int32_t *data2_g;
	int32_t *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (int32_t)i;
		data2[i] = (int32_t)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&data2_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(int32_t)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );

	max_flops<int32_t><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, data2_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );

	float latency;
	latency = ((float)(stopClk[0]-startClk[0]))/((float)(REPEAT_TIMES));
	printf("DPX latency = %f (clk)\n", latency);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 

