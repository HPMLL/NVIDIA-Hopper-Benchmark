#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>               // CUtensormap
#include <cuda/barrier>
#include "../../util.h"


using barrier = cuda::barrier<cuda::thread_scope_block>;

// float
typedef float dtype;
CUtensorMapDataType tm_dtype = CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32;


#define ARRAY_SIZE (4 * 1024*1024*(1024/sizeof(dtype))) // GB
#define GMEM_X 1024
#define GMEM_Y 1024
#define GMEM_Z 1024
// constexpr uint SMEM_X[] = {8,8,16,16,16,16};
// constexpr uint SMEM_Y[] = {8,8,8,16,16,16};
// constexpr uint SMEM_Z[] = {4,8,8,8,12,16};
constexpr uint SMEM_X[] = {16,64,256,16,4};
constexpr uint SMEM_Y[] = {16,64,16,256,4};
constexpr uint SMEM_Z[] = {16,1,1,1,256};
constexpr uint BLOCKS[] = {114, 228, 342, 456};	
#define THREADS_PER_BLOCK 1024
constexpr uint IDX = 4;
constexpr uint LOAD_SIZE = (SMEM_X[IDX] * SMEM_Y[IDX] * SMEM_Z[IDX] * sizeof(dtype)); //bytes


__global__ void init_data(dtype * array) {
    uint32_t tid = threadIdx.x;
	uint32_t uid = blockIdx.x * blockDim.x + tid;
    auto total_threads = blockDim.x * gridDim.x;

	for (uint32_t i = uid; i < ARRAY_SIZE; i += total_threads) {
		array[i] = uid;
    }
}

__global__ void tma_bw_3d(const __grid_constant__ CUtensorMap tma_desc, dtype *dsink)
{

    uint32_t tid = threadIdx.x;
	uint32_t uid = blockIdx.x * blockDim.x + tid;
    // dtype temp_res = 0;

    __shared__ alignas(16) dtype smem[LOAD_SIZE/sizeof(dtype)];

#pragma nv_diag_suppress static_var_with_dynamic_init
    __shared__ barrier bar;
    if (tid == 0) {
        init(&bar, blockDim.x);                    // a)
        asm volatile("fence.proxy.async.shared::cta;");     // b)
        
        for (int i = uid; i < ARRAY_SIZE * sizeof(dtype) / LOAD_SIZE; i += gridDim.x * blockDim.x) {
            int tensor_cood_x = (i % ((GMEM_X / SMEM_X[IDX]) * (GMEM_Y / SMEM_Y[IDX]))) % (GMEM_X / SMEM_X[IDX]) * SMEM_X[IDX];
            int tensor_cood_y = (i % ((GMEM_X / SMEM_X[IDX]) * (GMEM_Y / SMEM_Y[IDX])) / (GMEM_X / SMEM_X[IDX])) * SMEM_Y[IDX];
            int tensor_cood_z = i / ((GMEM_X / SMEM_X[IDX]) * (GMEM_Y / SMEM_Y[IDX])) * SMEM_Z[IDX];
            asm volatile(
                "{\t\n"
                //"discard.L2 [%1], 128;\n\t"
                "cp.async.bulk.tensor.3d.shared::cluster.global.tile.mbarrier::complete_tx::bytes [%0], [%1, {%2, %3, %4}], [%5]; // 1a. unicast\n\t"
                "mbarrier.expect_tx.relaxed.cta.shared::cta.b64 [%5], %6;\n\t"
                "}"
                :
                //: "r"(static_cast<unsigned>(__cvta_generic_to_shared(ptr))), "l"(ptr[0]), "n"(cuda::aligned_size_t<16>(LOAD_SIZE)), "r"(static_cast<unsigned>(__cvta_generic_to_shared(&bar)))
                : "r"(static_cast<unsigned>(__cvta_generic_to_shared(smem))), "l"(reinterpret_cast<uint64_t>(&tma_desc)), "r"(tensor_cood_x), "r"(tensor_cood_y), "r"(tensor_cood_z), "r"(static_cast<unsigned>(__cvta_generic_to_shared(&bar))), "n"(LOAD_SIZE)
                : "memory"); 


            // 3b. All threads arrive on the barrier
            barrier::arrival_token token = bar.arrive();

            // 3c. Wait for the data to have arrived.
            bar.wait(std::move(token));
            //temp_res += smem[0];
        }


    }


}

void create_tensor_map(CUtensorMap & tma_desc, dtype * array)
{
    auto rank = 3;
    uint64_t size[rank] = {GMEM_X, GMEM_Y, GMEM_Z};
    // The stride is the number of bytes to traverse from the first element of one row to the next.
    // It must be a multiple of 16.
    uint64_t stride[rank - 1] = {GMEM_X * sizeof(dtype), GMEM_X * sizeof(dtype) * GMEM_Y};
    // The box_size is the size of the shared memory buffer that is used as the destination of a TMA transfer.
    uint32_t box_size[rank] = {SMEM_X[IDX], SMEM_Y[IDX], SMEM_Z[IDX]};
    // The distance between elements in units of sizeof(element). A stride of 2
    // can be used to load only the real component of a complex-valued tensor, for instance.
    uint32_t elem_stride[rank] = {1, 1, 1};
    // Interleave patterns are sometimes used to accelerate loading of values that
    // are less than 4 bytes long.
    CUtensorMapInterleave interleave = CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE;
    // Swizzling can be used to avoid shared memory bank conflicts.
    CUtensorMapSwizzle swizzle = CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE;
    CUtensorMapL2promotion l2_promotion = CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE;
    // Any element that is outside of bounds will be set to zero by the TMA transfer.
    CUtensorMapFloatOOBfill oob_fill = CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE;

    // Get a function pointer to the cuTensorMapEncodeTiled driver API.
    auto cuTensorMapEncodeTiled = get_cuTensorMapEncodeTiled();

    // Create the tensor descriptor.
    hipError_t res = cuTensorMapEncodeTiled(
        &tma_desc,    // CUtensorMap *tensorMap,
        tm_dtype,        // CUtensorMapDataType tensorDataType,
        rank,         // cuuint32_t tensorRank,
        array,       // void *globalAddress,
        size,         // const cuuint64_t *globalDim,
        stride,       // const cuuint64_t *globalStrides,
        box_size,     // const cuuint32_t *boxDim,
        elem_stride,  // const cuuint32_t *elementStrides,
        interleave,   // CUtensorMapInterleave interleave,
        swizzle,      // CUtensorMapSwizzle swizzle,
        l2_promotion, // CUtensorMapL2promotion l2Promotion,
        oob_fill      // CUtensorMapFloatOOBfill oobFill);
    );
    printf("cuTensorMapEncodeTiled returned hipError_t: %d\n\n", res);

}

int main() {

 for (int i = 0; i < sizeof(BLOCKS)/sizeof(int); ++i) {
        printf("\nBlock size = %d, X = %d, Y = %d, Z = %d, Load size = %d KB\n", BLOCKS[i], SMEM_X[IDX], SMEM_Y[IDX], SMEM_Z[IDX], LOAD_SIZE/1024);
        dtype *dsink = (dtype *)malloc(sizeof(dtype));

        dtype *array_g;
        dtype *dsink_g;

        CUDA_CHECK(hipMalloc(&array_g, sizeof(dtype) * ARRAY_SIZE));
        CUDA_CHECK(hipMalloc(&dsink_g, sizeof(dtype)));

        init_data<<<BLOCKS[i], THREADS_PER_BLOCK>>>(array_g);

        CUtensorMap tma_desc{};
        create_tensor_map(tma_desc, array_g);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        tma_bw_3d<<<BLOCKS[i], 1>>>(tma_desc, dsink_g);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        CUDA_CHECK(hipPeekAtLastError());
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        CUDA_CHECK(hipMemcpy(dsink, dsink_g, sizeof(dtype), hipMemcpyDeviceToHost));
        printf("Total time = %f ms, transfer size = %lu bytes\n", milliseconds, ARRAY_SIZE * sizeof(dtype));
        printf("Throughput: %f GB/s\n\n", ARRAY_SIZE * sizeof(dtype) / (milliseconds / 1000) / 1024 / 1024 / 1024);
    }    
}