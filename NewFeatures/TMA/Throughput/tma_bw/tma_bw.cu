#include "hip/hip_runtime.h"
#include <cuda/barrier>
#include "../../util.h"


using barrier = cuda::barrier<cuda::thread_scope_block>;


typedef float dtype;

#define ARRAY_SIZE (4 * 1024*1024*(1024/sizeof(dtype))) // GB
constexpr int BLOCKS[] = {114, 228, 342, 456};
#define THREADS_PER_BLOCK 1024

constexpr int LOAD_SIZE_LIST[] = {1*1024, 2*1024, 4*1024, 8*1024, 12*1024, 16*1024}; //bytes
constexpr int LOAD_SIZE = LOAD_SIZE_LIST[2];

__global__ void init_data(dtype * array) {
    uint32_t tid = threadIdx.x;
	uint32_t uid = blockIdx.x * blockDim.x + tid;
    auto total_threads = blockDim.x * gridDim.x;

	for (uint32_t i = uid; i < ARRAY_SIZE; i += total_threads) {
		array[i] = uid;
    }
}

__global__ void tma_bw(dtype * volatile array, dtype *dsink)
{

    uint32_t tid = threadIdx.x;
	uint32_t uid = blockIdx.x * blockDim.x + tid;
    // dtype temp_res = 0;

    __shared__ alignas(16) dtype smem[LOAD_SIZE/sizeof(dtype)];

#pragma nv_diag_suppress static_var_with_dynamic_init
    __shared__ barrier bar;
    if (tid == 0) {
        init(&bar, blockDim.x);                    // a)
        asm volatile("fence.proxy.async.shared::cta;");     // b)
        
        for (int i = uid * (LOAD_SIZE / sizeof(dtype)); i < ARRAY_SIZE; i += gridDim.x * blockDim.x * (LOAD_SIZE / sizeof(dtype))) {

            auto ptr = array + i;

            asm volatile(
                "{\t\n"
                //"discard.L2 [%1], 128;\n\t"
                "cp.async.bulk.shared::cluster.global.mbarrier::complete_tx::bytes[%0], [%1], %2, [%3]; // 1a. unicast\n\t"
                "mbarrier.expect_tx.relaxed.cta.shared::cta.b64 [%3], %2;\n\t"
                "}"
                :
                //: "r"(static_cast<unsigned>(__cvta_generic_to_shared(ptr))), "l"(ptr[0]), "n"(cuda::aligned_size_t<16>(LOAD_SIZE)), "r"(static_cast<unsigned>(__cvta_generic_to_shared(&bar)))
                : "r"(static_cast<unsigned>(__cvta_generic_to_shared(smem))), "l"(ptr), "n"(LOAD_SIZE), "r"(static_cast<unsigned>(__cvta_generic_to_shared(&bar)))
                : "memory"); 


            // 3b. All threads arrive on the barrier
            barrier::arrival_token token = bar.arrive();

            // 3c. Wait for the data to have arrived.
            bar.wait(std::move(token));
            //temp_res += smem[0];
        }


    }


}

int main() {

    for (int i = 0; i < sizeof(BLOCKS)/sizeof(int); ++i) {
        printf("Block size = %d, Load size = %d KB\n", BLOCKS[i], LOAD_SIZE/1024);
        dtype *dsink = (dtype *)malloc(sizeof(dtype));

        dtype *array_g;
        dtype *dsink_g;

        CUDA_CHECK(hipMalloc(&array_g, sizeof(dtype) * ARRAY_SIZE));
        CUDA_CHECK(hipMalloc(&dsink_g, sizeof(dtype)));

        init_data<<<BLOCKS[i], THREADS_PER_BLOCK>>>(array_g);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        tma_bw<<<BLOCKS[i], 1>>>(array_g, dsink_g);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        CUDA_CHECK(hipPeekAtLastError());
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        CUDA_CHECK(hipMemcpy(dsink, dsink_g, sizeof(dtype), hipMemcpyDeviceToHost));
        printf("Total time = %f ms, transfer size = %lu bytes\n", milliseconds, ARRAY_SIZE * sizeof(dtype));
        printf("Throughput: %f GB/s\n", ARRAY_SIZE * sizeof(dtype) / (milliseconds / 1000) / 1024 / 1024 / 1024);
    }

    
}