#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define THREADS_PER_BLOCK 1
#define THREADS_PER_SM 1
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 4096

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


// template <class T>
// __global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
// 	int gid = blockIdx.x*blockDim.x + threadIdx.x;
// 	register T s1 = data1[gid];
// 	register T s2 = data2[gid];
// 	register T result = 0;

// 	// synchronize all threads
// 	asm volatile ("bar.sync 0;");

// 	// start timing
// 	uint32_t start = 0;
// 	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

// 	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
// 		asm volatile ("{\t\n"
// 				"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 				"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 				"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 				"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 				"}" : "+f"(result),"+f"(s1),"+f"(s2)
// 		);

// 	}
// 	// synchronize all threads
// 	asm volatile("bar.sync 0;");

// 	// stop timing
// 	uint32_t stop = 0;
// 	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

// 	// write time and data back to memory
// 	startClk[gid] = start;

// 	stopClk[gid] = stop;
// 	res[gid] = result;
// }

template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	register T s1 = data1[gid];
	register T s2 = data2[gid];
	register T s3 = s1;
	register T s4 = s2;
	register T result = 1;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		// asm volatile ("{\t\n"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"}" : "+f"(result),"+f"(s1),"+f"(s2)
		// );

		s1 += s1 * s2;
		s2 += s2 * s3;
		s3 += s3 * s4;
		s4 += s4 * s1;

	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	result = s1 + s2 + s3 + s4;
	// write time and data back to memory
	startClk[gid] = start;

	stopClk[gid] = stop;
	res[gid] = result;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	half *data1 = (half*) malloc(TOTAL_THREADS*sizeof(half));
	half *data2 = (half*) malloc(TOTAL_THREADS*sizeof(half));
	half *res = (half*) malloc(TOTAL_THREADS*sizeof(half));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	half *data1_g;
	half *data2_g;
	half *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (half)i;
		data2[i] = (half)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(half)) );
	gpuErrchk( hipMalloc(&data2_g, TOTAL_THREADS*sizeof(half)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(half)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(half), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(half), hipMemcpyHostToDevice) );

	max_flops<half><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, data2_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(half), hipMemcpyDeviceToHost) );


	float latency;
	latency = ((float)(stopClk[0]-startClk[0]))/((float)(REPEAT_TIMES*4));
	printf("fp16 latency = %f (clk)\n", latency);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 

