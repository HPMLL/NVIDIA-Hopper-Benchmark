#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <algorithm>

#define THREADS_PER_BLOCK 1024
#define THREADS_PER_SM 1024
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 1024

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


// template <class T>
// __global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
// 	int gid = blockIdx.x*blockDim.x + threadIdx.x;
// 	register T s1 = data1[gid];
// 	register T s2 = data2[gid];
// 	register T result = 0;

// 	// synchronize all threads
// 	asm volatile ("bar.sync 0;");

// 	// start timing
// 	uint32_t start = 0;
// 	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

// 	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
// 		asm volatile ("{\t\n"
// 				"mad.lo.s32 %0, %1, %2 , %0;\n\t"
// 				"mad.lo.s32 %0, %1, %2 , %0;\n\t"
// 				"mad.lo.s32 %0, %1, %2 , %0;\n\t"
// 				"mad.lo.s32 %0, %1, %2 , %0;\n\t"
// 				"}" : "+r"(result),"+r"(s1),"+r"(s2)
// 		);



// 	}
// 	// synchronize all threads
// 	asm volatile("bar.sync 0;");

// 	// stop timing
// 	uint32_t stop = 0;
// 	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

// 	// write time and data back to memory
// 	startClk[gid] = start;
// 	stopClk[gid] = stop;
// 	res[gid] = result;
// }

template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	register T s1 = data1[gid];
	register T s2 = data2[gid];
	register T s3 = s1;
	register T s4 = s2;
	register T result = 1;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		// asm volatile ("{\t\n"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
		// 		"}" : "+f"(result),"+f"(s1),"+f"(s2)
		// );

		s1 += s1 * s2;
		s2 += s2 * s3;
		s3 += s3 * s4;
		s4 += s4 * s1;

	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	result = s1 + s2 + s3 + s4;
	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = result;
}


int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	int32_t *data1 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *data2 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *res = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	int32_t *data1_g;
	int32_t *data2_g;
	int32_t *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (int32_t)i;
		data2[i] = (int32_t)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&data2_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(int32_t)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );

	max_flops<int32_t><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, data2_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );

	auto dstart = *std::min_element(&startClk[0],&startClk[TOTAL_THREADS]);
	auto dend = *std::max_element(&stopClk[0],&stopClk[TOTAL_THREADS]);
	auto total_time = dend - dstart;

	float flops;
	flops = (float)(REPEAT_TIMES*THREADS_PER_SM*8)/((float)total_time);
	printf("FLOP per SM = %f (flop/clk/SM)\n", flops);
	printf("Total Clk number = %u \n", total_time);

	return 0;
} 

