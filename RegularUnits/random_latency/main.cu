#include "hip/hip_runtime.h"
#include "MeasurementSeries.hpp"
#include "dtime.hpp"
#include "gpu-clock.cuh"
#include "gpu-error.h"
#include <algorithm>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <omp.h>
#include <random>
#include <sys/time.h>

using namespace std;

__device__ unsigned int smid() {
  unsigned int r;

  asm("mov.u32 %0, %%smid;" : "=r"(r));

  return r;
}

template <typename T>
__global__ void pchase(T *buf, T *__restrict__ dummy_buf, int64_t N) {

  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t *idx = buf;

  const int unroll_factor = 32;
#pragma unroll 1
  for (int64_t n = 0; n < N; n += unroll_factor) {
#pragma unroll
    for (int u = 0; u < unroll_factor; u++) {
      idx = (int64_t *)*idx;
    }
  }

  if (tidx > 12313) {
    dummy_buf[0] = (int64_t)idx;
  }
}

int main(int argc, char **argv) {
  int device = 0;
  if (argc > 1) {
    device = atoi(argv[1]);
    hipSetDevice(device);
  }
  unsigned int clock = getGPUClock(device);

  typedef int64_t dtype;

  const int cl_size = 1;
  const int skip_factor = 16;

  for (int64_t LEN = 16; LEN < (1 << 24); LEN = LEN * 1.04 + 32) {
    if (LEN * skip_factor * cl_size * sizeof(dtype) > 120 * 1024 * 1024)
      LEN *= 1.5;

    const int64_t iters = max(LEN, (int64_t)1000000);
    // const int64_t iters =
    //     max((int64_t)2, ((int64_t)1 << 19) / LEN) * LEN * cl_size;

    vector<int64_t> order(LEN);
    int64_t *buf = NULL;
    int64_t *dbuf = NULL;
    dtype *dummy_buf = NULL;

    GPU_ERROR(
        hipMallocManaged(&buf, skip_factor * cl_size * LEN * sizeof(dtype)));
    GPU_ERROR(hipMalloc(&dbuf, skip_factor * cl_size * LEN * sizeof(dtype)));
    GPU_ERROR(hipMallocManaged(&dummy_buf, sizeof(dtype)));
    for (int64_t i = 0; i < LEN; i++) {
      order[i] = i + 1;
    }
    order[LEN - 1] = 0;

    std::random_device rd;
    std::mt19937 g(rd());
    shuffle(begin(order), end(order) - 1, g);

    for (int cl_lane = 0; cl_lane < cl_size; cl_lane++) {
      dtype idx = 0;
      for (int64_t i = 0; i < LEN; i++) {

        buf[(idx * cl_size + cl_lane) * skip_factor] =
            skip_factor *
            (order[i] * cl_size + cl_lane + (order[i] == 0 ? 1 : 0));
        idx = order[i];
      }
    }
    buf[skip_factor * (order[LEN - 2] * cl_size + cl_size - 1)] = 0;

    for (int64_t n = 0; n < LEN * cl_size * skip_factor; n++) {
      buf[n] = (int64_t)dbuf + buf[n] * sizeof(int64_t *);
    }

    hipMemcpy(dbuf, buf, skip_factor * cl_size * LEN * sizeof(dtype),
               hipMemcpyHostToDevice);

    pchase<dtype><<<1, 4>>>(dbuf, dummy_buf, iters);

    MeasurementSeries times;
    for (int i = 0; i < 7; i++) {
      GPU_ERROR(hipDeviceSynchronize());
      double start = dtime();
      pchase<dtype><<<1, 4>>>(dbuf, dummy_buf, iters);
      GPU_ERROR(hipDeviceSynchronize());
      double end = dtime();
      times.add(end - start);
    }

    GPU_ERROR(hipGetLastError());

    double dt = times.minValue();
    cout << setw(9) << iters << " " << setw(5) << clock << " " //
         << setw(8) << skip_factor * LEN * cl_size * sizeof(dtype) / 1024
         << " "                                            //
         << fixed                                          //
         << setprecision(1) << setw(8) << dt * 1000 << " " //
         << setw(7) << setprecision(1)
         << (double)dt / iters * clock * 1000 * 1000 << "\n"
         << flush;

    GPU_ERROR(hipFree(buf));
    GPU_ERROR(hipFree(dbuf));
    GPU_ERROR(hipFree(dummy_buf));
  }
  cout << "\n";
}
