#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1
#define THREADS_PER_SM 1
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 4096

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	register T s1 = data1[gid];
	register T s2 = data2[gid];
	register T result = 0;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		asm volatile ("{\t\n"
				"fma.rn.f64 %0, %1, %2 , %0;\n\t"
				"fma.rn.f64 %0, %1, %2 , %0;\n\t"
				"fma.rn.f64 %0, %1, %2 , %0;\n\t"
				"fma.rn.f64 %0, %1, %2 , %0;\n\t"
				"}" : "+d"(result),"+d"(s1),"+d"(s2)
		);



	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = result;
}

// template <class T>
// __global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
// 	int gid = blockIdx.x*blockDim.x + threadIdx.x;
// 	register T s1 = data1[gid];
// 	register T s2 = data2[gid];
// 	register T s3 = s1;
// 	register T s4 = s2;
// 	register T result = 1;

// 	// synchronize all threads
// 	asm volatile ("bar.sync 0;");

// 	// start timing
// 	uint32_t start = 0;
// 	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

// 	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
// 		// asm volatile ("{\t\n"
// 		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 		// 		"fma.rn.f32 %0, %1, %2 , %0;\n\t"
// 		// 		"}" : "+f"(result),"+f"(s1),"+f"(s2)
// 		// );

// 		s1 += s1 * s2;
// 		s2 += s2 * s3;
// 		s3 += s3 * s4;
// 		s4 += s4 * s1;

// 	}
// 	// synchronize all threads
// 	asm volatile("bar.sync 0;");

// 	// stop timing
// 	uint32_t stop = 0;
// 	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

// 	result = s1 + s2 + s3 + s4;
// 	// write time and data back to memory
// 	startClk[gid] = start;
// 	stopClk[gid] = stop;
// 	res[gid] = result;
// }

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	double *data1 = (double*) malloc(TOTAL_THREADS*sizeof(double));
	double *data2 = (double*) malloc(TOTAL_THREADS*sizeof(double));
	double *res = (double*) malloc(TOTAL_THREADS*sizeof(double));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	double *data1_g;
	double *data2_g;
	double *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (double)i;
		data2[i] = (double)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(double)) );
	gpuErrchk( hipMalloc(&data2_g, TOTAL_THREADS*sizeof(double)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(double)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(double), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(double), hipMemcpyHostToDevice) );

	max_flops<double><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, data2_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(double), hipMemcpyDeviceToHost) );

	float latency;
	latency = ((float)(stopClk[0]-startClk[0]))/((float)(REPEAT_TIMES*4));
	printf("fp64 latency = %f (clk)\n", latency);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 

